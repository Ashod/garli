#include "hip/hip_runtime.h"
/*
 * cudakernel.cu
 *
 *  Created on: Nov 12, 2008
 *      Author: ayres
 */

#ifndef CUDAKERNELS
#define CUDAKERNELS

__global__ void GarliCLANucleotideNRate(FLOAT_TYPE* CLA, FLOAT_TYPE* Lpr,
		FLOAT_TYPE* Rpr, FLOAT_TYPE* LCL, FLOAT_TYPE* RCL, int nRateCats) {
	// has to be run on blocks of size RATES*4*16, one dimension

	// Block index pre-multiplied
	int bxpm = blockIdx.x * nRateCats * 64;

	// Thread index
	int tx = threadIdx.x;

	// CLAsubL and R are used to store the left and right elements of the block sub-matrix that are computed by the thread
	float CLAsubL = 0;
	float CLAsubR = 0;

	// Declaration of the shared memory array Lprs and Rprs used to store the sub-matrix of Lpr and Rpr
	__shared__ float Lprs[4*16];
	__shared__ float Rprs[4*16];

	// Declaration of the shared memory array LCLs and RCLs used to store the sub-matrix of LCL and RCL
	__shared__ float LCLs[4*4*16];
	__shared__ float RCLs[4*4*16];

	// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
	if (tx < 64) {
		Lprs[tx] = Lpr[tx];
		Rprs[tx] = Rpr[tx];
	}

	// it's not clear to me why this needs to be here but the 1 rate case gives errors without it
	__syncthreads();

	LCLs[tx] = LCL[bxpm + tx];
	RCLs[tx] = RCL[bxpm + tx];

	// Synchronize to make sure the matrices are loaded
	__syncthreads();

	// Multiply the two matrices together, each thread computes one element of the block sub-matrix
	for (int k = 0; k < 4; ++k) {
		CLAsubL += Lprs[4 * (tx & (nRateCats * 4 - 1)) + k] * LCLs[tx - (tx
				& (3)) + k];
		CLAsubR += Rprs[4 * (tx & (nRateCats * 4 - 1)) + k] * RCLs[tx - (tx
				& (3)) + k];
	}

	// Write the block sub-matrix to device memory each thread writes one element
	CLA[bxpm + tx] = CLAsubL * CLAsubR;
}

__global__ void GarliCLAAminoAcidNRate(FLOAT_TYPE* CLA, FLOAT_TYPE* Lpr,
		FLOAT_TYPE* Rpr, FLOAT_TYPE* LCL, FLOAT_TYPE* RCL, int nRateCats) {
	// Block size has to be 20x20

	// Declaration of the shared memory array prs used to store the sub-matrix of Lpr and Rpr
	__shared__ float prs[20][20];

	// Declaration of the shared memory array CLs used to store the sub-matrix of LCL and RCL
	__shared__ float CLs[20][20];

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Block index
	int CLBlockIdx = blockIdx.x * nRateCats * 400;

	for (int i = 0; i < nRateCats; ++i) {

		int prIdx = i * 400 + ty * 20 + tx;
		int CLIdx = i * 20 + nRateCats * 20 * ty + tx;

		float CLAsubL = 0;
		float CLAsubR = 0;

		__syncthreads();

		// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
		prs[tx][ty] = Lpr[prIdx];
		CLs[ty][tx] = LCL[CLBlockIdx + CLIdx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together, each thread computes one element of the block sub-matrix
		for (int k = 0; k < 20; ++k)
			CLAsubL += prs[k][tx] * CLs[ty][k];

		// Synchronize to make sure the L side is done
		__syncthreads();

		// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
		prs[tx][ty] = Rpr[prIdx];
		CLs[ty][tx] = RCL[CLBlockIdx + CLIdx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together, each thread computes one element of the block sub-matrix
		for (int k = 0; k < 20; ++k)
			CLAsubR += prs[k][tx] * CLs[ty][k];

		// Write the block sub-matrix to device memory each thread writes one element
		CLA[CLIdx + CLBlockIdx] = CLAsubL * CLAsubR;
	}
}

__global__ void GarliCLACodonNRate(FLOAT_TYPE* CLA, FLOAT_TYPE* Lpr,
		FLOAT_TYPE* Rpr, FLOAT_TYPE* LCL, FLOAT_TYPE* RCL, int nRateCats) {
	// Block size has to be 16x16

	// Declaration of the shared memory array prs used to store the sub-matrix of Lpr and Rpr
	__shared__ float prs[16][16];

	// Declaration of the shared memory array CLs used to store the sub-matrix of LCL and RCL
	__shared__ float CLs[16][16];

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Block indexes
	int prBlockIdx = blockIdx.y * 61 * 16;
	int CLBlockIdx = blockIdx.x * nRateCats * 61 * 16;

	for (int i = 0; i < nRateCats; ++i) {

		int prIdx = i * 61 * 61 + ty * 61 + tx;
		int CLIdx = i * 61 + nRateCats * 61 * ty + tx;

		float CLAsubL = 0;
		float CLAsubR = 0;

		int j;
		for (j = 0; j < 48; j += 16) {

			// Synchronize before next round
			__syncthreads();

			// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
			CLs[ty][tx] = LCL[CLBlockIdx + CLIdx + j];
			prs[tx][ty] = Lpr[prBlockIdx + prIdx + j];

			// Synchronize to make sure the matrices are loaded
			__syncthreads();

			// Multiply the two matrices together, each thread computes one element of the block sub-matrix
			for (int k = 0; k < 16; ++k) {
				CLAsubL += prs[k][tx] * CLs[ty][k];
			}

			// Synchronize to make sure the L side is done
			__syncthreads();

			// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
			CLs[ty][tx] = RCL[CLBlockIdx + CLIdx + j];
			prs[tx][ty] = Rpr[prBlockIdx + prIdx + j];

			// Synchronize to make sure the matrices are loaded
			__syncthreads();

			// Multiply the two matrices together, each thread computes one element of the block sub-matrix
			for (int k = 0; k < 16; ++k) {
				CLAsubR += prs[k][tx] * CLs[ty][k];
			}
		}

		// For the last j step k only goes to 13 (61 states)
		__syncthreads();
		CLs[ty][tx] = LCL[CLBlockIdx + CLIdx + j];
		prs[tx][ty] = Lpr[prBlockIdx + prIdx + j];
		__syncthreads();
		for (int k = 0; k < 13; ++k) {
			CLAsubL += prs[k][tx] * CLs[ty][k];
		}
		__syncthreads();
		CLs[ty][tx] = RCL[CLBlockIdx + CLIdx + j];
		prs[tx][ty] = Rpr[prBlockIdx + prIdx + j];
		__syncthreads();
		for (int k = 0; k < 13; ++k) {
			CLAsubR += prs[k][tx] * CLs[ty][k];
		}

		// Write the block sub-matrix to device memory each thread writes one element
		if (blockIdx.y != 3 || tx < 13)
			CLA[CLIdx + (blockIdx.y * 16) + CLBlockIdx] = CLAsubL * CLAsubR;
	}
}

__global__ void GarliDerivNucleotideNRate(FLOAT_TYPE* d_partial,
		FLOAT_TYPE* d_CL1, int* d_partial_underflow_mult,
		int* d_CL1_underflow_mult, FLOAT_TYPE* d_prmat, FLOAT_TYPE* d_d1mat,
		FLOAT_TYPE* d_d2mat, FLOAT_TYPE* d_rateProb, FLOAT_TYPE* d_freqs,
		int* d_countit, int* d_conStates, FLOAT_TYPE* d_Tots_arr,
		int lastConst, bool NoPinvInModel, FLOAT_TYPE prI, int nRateCats) {
	__shared__ float prmat[4*4];
	__shared__ float d1mat[4*4];
	__shared__ float d2mat[4*4];
	__shared__ float CL1[4*128];
	__shared__ float freqs[4];
	__shared__ float rateProb[4];
	__shared__ float Tots[3*128];

	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int dimx = blockDim.x;
	int nstates = 4;
	int i = bx * blockDim.x + tx;

	if (tx < nstates)
		freqs[tx] = d_freqs[tx];
	else if (tx < (nstates + nRateCats))
		rateProb[tx - nstates] = d_rateProb[tx - nstates];

	// calculate remaining chars

	float siteL = 0, siteD1 = 0, siteD2 = 0;
	float tempL, tempD1, tempD2;
	float rateL, rateD1, rateD2;

	d_partial += nstates * nRateCats * i;
	d_CL1 += nstates * nRateCats * i;

	for (int rate = 0; rate < nRateCats; rate++) {
		rateL = rateD1 = rateD2 = 0;
		int rateOffset = rate * nstates * nstates;

		__syncthreads();
		if (tx < 16)
			prmat[tx] = d_prmat[rateOffset + tx];
		else if (tx < 32)
			d1mat[tx - 16] = d_d1mat[rateOffset + tx - 16];
		else if (tx < 48)
			d2mat[tx - 32] = d_d2mat[rateOffset + tx - 32];

		int from = 0;
		tempL = tempD1 = tempD2 = 0;
		for (int to = 0; to < nstates; to++) {
			CL1[nstates * tx + to] = d_CL1[to];
			__syncthreads();
			tempL += prmat[to] * CL1[nstates * tx + to];
			tempD1 += d1mat[to] * CL1[nstates * tx + to];
			tempD2 += d2mat[to] * CL1[nstates * tx + to];
		}
		float partial = d_partial[from];
		rateL += tempL * partial * freqs[from];
		rateD1 += tempD1 * partial * freqs[from];
		rateD2 += tempD2 * partial * freqs[from];

		for (from = 1; from < nstates; from++) {
			tempL = tempD1 = tempD2 = 0;
			int offset = from * nstates;
			for (int to = 0; to < nstates; to++) {
				tempL += prmat[offset + to] * CL1[nstates * tx + to];
				tempD1 += d1mat[offset + to] * CL1[nstates * tx + to];
				tempD2 += d2mat[offset + to] * CL1[nstates * tx + to];
			}
			partial = d_partial[from];
			rateL += tempL * partial * freqs[from];
			rateD1 += tempD1 * partial * freqs[from];
			rateD2 += tempD2 * partial * freqs[from];
		}
		siteL += rateL * rateProb[rate];
		siteD1 += rateD1 * rateProb[rate];
		siteD2 += rateD2 * rateProb[rate];

		d_partial += nstates;
		d_CL1 += nstates;
	}

	float partial_underflow_mult = d_partial_underflow_mult[i];
	float CL1_underflow_mult = d_CL1_underflow_mult[i];

	if ((NoPinvInModel == false) && (i <= lastConst)) {
		float btot = 0.0f;
		int conStates = d_conStates[i];
		if (conStates & 1)
			btot += freqs[0];
		if (conStates & 2)
			btot += freqs[1];
		if (conStates & 4)
			btot += freqs[2];
		if (conStates & 8)
			btot += freqs[3];
		siteL += (prI * btot)
				* exp(partial_underflow_mult + CL1_underflow_mult);
	}

	int countit = d_countit[i];

	Tots[tx] = (log(siteL) - partial_underflow_mult - CL1_underflow_mult)
			* countit;
	siteD1 /= siteL;
	Tots[tx + dimx] = countit * siteD1;
	Tots[tx + dimx * 2] = countit * ((siteD2 / siteL) - siteD1 * siteD1);
	__syncthreads();
	for (unsigned int s = dimx / 2; s > 0; s >>= 1) {
		if (tx < s) {
			Tots[tx] += Tots[tx + s];
			Tots[tx + dimx] += Tots[tx + dimx + s];
			Tots[tx + dimx * 2] += Tots[tx + dimx * 2 + s];
		}
		__syncthreads();
	}
	if (tx == 0) {
		d_Tots_arr[bx] = Tots[0];
		d_Tots_arr[bx + gridDim.x] = Tots[0 + dimx];
		d_Tots_arr[bx + gridDim.x * 2] = Tots[0 + dimx * 2];
	}

}

__global__ void GarliDerivAminoAcidNRate(FLOAT_TYPE* d_partial,
		FLOAT_TYPE* d_CL1, int* d_partial_underflow_mult,
		int* d_CL1_underflow_mult, FLOAT_TYPE* d_prmat, FLOAT_TYPE* d_d1mat,
		FLOAT_TYPE* d_d2mat, FLOAT_TYPE* d_rateProb, FLOAT_TYPE* d_freqs,
		int* d_countit, int* d_conStates, FLOAT_TYPE* d_Tots_arr,
		int lastConst, bool NoPinvInModel, FLOAT_TYPE prI, int nRateCats) {
	__shared__ float prmat[20*20];
	__shared__ float d1mat[20*20];
	__shared__ float d2mat[20*20];
	__shared__ float CL1[20*128];
	__shared__ float freqs[20];
	__shared__ float rateProb[4];
	__shared__ float Tots[128];

	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int dimx = blockDim.x;
	int nstates = 20;
	int i = bx * blockDim.x + tx;

	if (tx < nstates)
		freqs[tx] = d_freqs[tx];
	else if (tx < (nstates + nRateCats))
		rateProb[tx - nstates] = d_rateProb[tx - nstates];

	// calculate remaining chars

	float siteL = 0, siteD1 = 0, siteD2 = 0;
	float tempL, tempD1, tempD2;
	float rateL, rateD1, rateD2;

	d_partial += nstates * nRateCats * i;
	d_CL1 += nstates * nRateCats * i;

	for (int rate = 0; rate < nRateCats; rate++) {
		rateL = rateD1 = rateD2 = 0;
		int rateOffset = rate * nstates * nstates;

		__syncthreads();
		for (int j = 0; j < 3; j++) {
			prmat[tx + dimx * j] = d_prmat[rateOffset + tx + dimx * j];
			d1mat[tx + dimx * j] = d_d1mat[rateOffset + tx + dimx * j];
			d2mat[tx + dimx * j] = d_d2mat[rateOffset + tx + dimx * j];
		}
		if (tx < 16)
			prmat[tx + dimx * 3] = d_prmat[rateOffset + tx + dimx * 3];
		else if (tx < 32)
			d1mat[tx - 16 + dimx * 3]
					= d_d1mat[rateOffset + tx - 16 + dimx * 3];
		else if (tx < 48)
			d2mat[tx - 32 + dimx * 3]
					= d_d2mat[rateOffset + tx - 32 + dimx * 3];

		int from = 0;
		tempL = tempD1 = tempD2 = 0;
		for (int to = 0; to < nstates; to++) {
			CL1[nstates * tx + to] = d_CL1[to];
			__syncthreads();
			tempL += prmat[to] * CL1[nstates * tx + to];
			tempD1 += d1mat[to] * CL1[nstates * tx + to];
			tempD2 += d2mat[to] * CL1[nstates * tx + to];
		}
		float partial = d_partial[from];
		rateL += tempL * partial * freqs[from];
		rateD1 += tempD1 * partial * freqs[from];
		rateD2 += tempD2 * partial * freqs[from];

		for (from = 1; from < nstates; from++) {
			tempL = tempD1 = tempD2 = 0;
			int offset = from * nstates;
			for (int to = 0; to < nstates; to++) {
				tempL += prmat[offset + to] * CL1[nstates * tx + to];
				tempD1 += d1mat[offset + to] * CL1[nstates * tx + to];
				tempD2 += d2mat[offset + to] * CL1[nstates * tx + to];
			}
			partial = d_partial[from];
			rateL += tempL * partial * freqs[from];
			rateD1 += tempD1 * partial * freqs[from];
			rateD2 += tempD2 * partial * freqs[from];
		}
		siteL += rateL * rateProb[rate];
		siteD1 += rateD1 * rateProb[rate];
		siteD2 += rateD2 * rateProb[rate];

		d_partial += nstates;
		d_CL1 += nstates;
	}

	float partial_underflow_mult = d_partial_underflow_mult[i];
	float CL1_underflow_mult = d_CL1_underflow_mult[i];

	if ((NoPinvInModel == false) && (i <= lastConst))
		siteL += (prI * freqs[d_conStates[i]] * exp(partial_underflow_mult
				+ CL1_underflow_mult));

	int countit = d_countit[i];

	Tots[tx] = (log(siteL) - partial_underflow_mult - CL1_underflow_mult)
			* countit;
	__syncthreads();
	for (unsigned int s = dimx / 2; s > 0; s >>= 1) {
		if (tx < s)
			Tots[tx] += Tots[tx + s];
		__syncthreads();
	}
	if (tx == 0)
		d_Tots_arr[bx] = Tots[0];

	siteD1 /= siteL;
	Tots[tx] = countit * siteD1;
	__syncthreads();
	for (unsigned int s = dimx / 2; s > 0; s >>= 1) {
		if (tx < s)
			Tots[tx] += Tots[tx + s];
		__syncthreads();
	}
	if (tx == 0)
		d_Tots_arr[bx + gridDim.x] = Tots[0];

	Tots[tx] = countit * ((siteD2 / siteL) - siteD1 * siteD1);
	__syncthreads();
	for (unsigned int s = dimx / 2; s > 0; s >>= 1) {
		if (tx < s)
			Tots[tx] += Tots[tx + s];
		__syncthreads();
	}
	if (tx == 0)
		d_Tots_arr[bx + gridDim.x * 2] = Tots[0];

}

__global__ void GarliDerivCodonNRate(FLOAT_TYPE* d_partial, FLOAT_TYPE* d_CL1,
		int* d_partial_underflow_mult, int* d_CL1_underflow_mult,
		FLOAT_TYPE* d_prmat, FLOAT_TYPE* d_d1mat, FLOAT_TYPE* d_d2mat,
		FLOAT_TYPE* d_rateProb, FLOAT_TYPE* d_freqs, int* d_countit,
		int* d_conStates, FLOAT_TYPE* d_Tots_arr, int lastConst,
		bool NoPinvInModel, FLOAT_TYPE prI, int nRateCats) {
	__shared__ float freqs[61];
	__shared__ float rateProb[4];
	__shared__ float Tots[3*128];

	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int dimx = blockDim.x;
	int nstates = 61;
	int i = bx * blockDim.x + tx;

	if (tx < nstates)
		freqs[tx] = d_freqs[tx];
	else if (tx < (nstates + nRateCats))
		rateProb[tx - nstates] = d_rateProb[tx - nstates];

	__syncthreads();

	// calculate remaining chars

	float siteL = 0, siteD1 = 0, siteD2 = 0;
	float tempL, tempD1, tempD2;
	float rateL, rateD1, rateD2;

	d_partial += nstates * nRateCats * i;
	d_CL1 += nstates * nRateCats * i;

	for (int rate = 0; rate < nRateCats; rate++) {
		rateL = rateD1 = rateD2 = 0;
		int rateOffset = rate * nstates * nstates;

		for (int from = 0; from < nstates; from++) {
			tempL = tempD1 = tempD2 = 0;
			int offset = from * nstates;
			for (int to = 0; to < nstates; to++) {
				float CL1 = d_CL1[to];
				tempL += d_prmat[rateOffset + offset + to] * CL1;
				tempD1 += d_d1mat[rateOffset + offset + to] * CL1;
				tempD2 += d_d2mat[rateOffset + offset + to] * CL1;
			}
			float partial = d_partial[from];
			rateL += tempL * partial * freqs[from];
			rateD1 += tempD1 * partial * freqs[from];
			rateD2 += tempD2 * partial * freqs[from];
		}
		siteL += rateL * rateProb[rate];
		siteD1 += rateD1 * rateProb[rate];
		siteD2 += rateD2 * rateProb[rate];

		d_partial += nstates;
		d_CL1 += nstates;
	}

	float partial_underflow_mult = d_partial_underflow_mult[i];
	float CL1_underflow_mult = d_CL1_underflow_mult[i];

	if ((NoPinvInModel == false) && (i <= lastConst))
		siteL += (prI * freqs[d_conStates[i]] * exp(partial_underflow_mult
				+ CL1_underflow_mult));

	int countit = d_countit[i];

	Tots[tx] = (log(siteL) - partial_underflow_mult - CL1_underflow_mult)
			* countit;
	siteD1 /= siteL;
	Tots[tx + dimx] = countit * siteD1;
	Tots[tx + dimx * 2] = countit * ((siteD2 / siteL) - siteD1 * siteD1);
	__syncthreads();
	for (unsigned int s = dimx / 2; s > 0; s >>= 1) {
		if (tx < s) {
			Tots[tx] += Tots[tx + s];
			Tots[tx + dimx] += Tots[tx + dimx + s];
			Tots[tx + dimx * 2] += Tots[tx + dimx * 2 + s];
		}
		__syncthreads();
	}
	if (tx == 0) {
		d_Tots_arr[bx] = Tots[0];
		d_Tots_arr[bx + gridDim.x] = Tots[0 + dimx];
		d_Tots_arr[bx + gridDim.x * 2] = Tots[0 + dimx * 2];
	}
}

#endif// #ifndef CUDAKERNELS
