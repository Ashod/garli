#include "hip/hip_runtime.h"
/*
 * cudainterface.cu
 *
 *  Created on: Nov 12, 2008
 *      Author: ayres
 */


#include <cutil.h>

#include "defs.h"
#include "outputman.h"

// includes, kernels
#include "cudakernel.cu"

extern OutputManager outman;

extern "C"
bool CheckCuda() {
    int deviceCount;
    bool cuda_support = false;

    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0)
    	outman.UserMessageNoCR("There is no device supporting CUDA\n");
    else
    	cuda_support = true;

    return cuda_support;
}

extern "C"
void DeviceQuery() {

	outman.UserMessageNoCR(
			"========================= GPU Device Query =========================\n");

    int deviceCount;
    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0)
    	outman.UserMessageNoCR("There is no device supporting CUDA\n");
    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999)
            	outman.UserMessageNoCR("There is no device supporting CUDA.\n");
            else if (deviceCount == 1)
            	outman.UserMessageNoCR("There is 1 device supporting CUDA\n");
            else
            	outman.UserMessageNoCR("There are %d devices supporting CUDA\n", deviceCount);
        }
        outman.UserMessageNoCR("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        outman.UserMessageNoCR("  Major revision number:                         %d\n",
               deviceProp.major);
        outman.UserMessageNoCR("  Minor revision number:                         %d\n",
               deviceProp.minor);
        outman.UserMessageNoCR("  Total amount of global memory:                 %u bytes\n",
               deviceProp.totalGlobalMem);
    #if CUDART_VERSION >= 2000
        outman.UserMessageNoCR("  Number of multiprocessors:                     %d\n",
               deviceProp.multiProcessorCount);
        outman.UserMessageNoCR("  Number of cores:                               %d\n",
               8 * deviceProp.multiProcessorCount);
    #endif
        outman.UserMessageNoCR("  Total amount of constant memory:               %u bytes\n",
               deviceProp.totalConstMem);
        outman.UserMessageNoCR("  Total amount of shared memory per block:       %u bytes\n",
               deviceProp.sharedMemPerBlock);
        outman.UserMessageNoCR("  Total number of registers available per block: %d\n",
               deviceProp.regsPerBlock);
        outman.UserMessageNoCR("  Warp size:                                     %d\n",
               deviceProp.warpSize);
        outman.UserMessageNoCR("  Maximum number of threads per block:           %d\n",
               deviceProp.maxThreadsPerBlock);
        outman.UserMessageNoCR("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        outman.UserMessageNoCR("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        outman.UserMessageNoCR("  Maximum memory pitch:                          %u bytes\n",
               deviceProp.memPitch);
        outman.UserMessageNoCR("  Texture alignment:                             %u bytes\n",
               deviceProp.textureAlignment);
        outman.UserMessageNoCR("  Clock rate:                                    %.2f GHz\n",
               deviceProp.clockRate * 1e-6f);
    #if CUDART_VERSION >= 2000
        outman.UserMessageNoCR("  Concurrent copy and execution:                 %s\n",
               deviceProp.deviceOverlap ? "Yes" : "No");
    #endif
    }

	outman.UserMessageNoCR(
				"====================================================================\n\n");
}


extern "C"
void SetDevice(unsigned int device_number) {
	hipSetDevice(device_number);
	    hipDeviceProp_t deviceProp;
	    hipGetDeviceProperties(&deviceProp, device_number);
	    outman.UserMessageNoCR ("Using GPU device %d: %s\n\n", device_number,deviceProp.name);
}

extern "C"
void AllocatePinnedMemory(void** arr, unsigned int mem_size_bytes) {

	// allocate host pinned memory
	CUDA_SAFE_CALL(hipHostMalloc((void**)&(*arr), mem_size_bytes));

}

extern "C"
void AllocateGPU(void** arr, unsigned int mem_size_bytes) {
	// allocate device memory
	CUDA_SAFE_CALL(hipMalloc((void**) &(*arr), mem_size_bytes));
}

extern "C"
void CuComputeGPUCLA(FLOAT_TYPE* h_Lpr, FLOAT_TYPE* h_Rpr, FLOAT_TYPE* h_LCL, FLOAT_TYPE* h_RCL, FLOAT_TYPE* h_CLA,
		FLOAT_TYPE* d_Lpr, FLOAT_TYPE* d_Rpr, FLOAT_TYPE* d_LCL, FLOAT_TYPE* d_RCL, FLOAT_TYPE* d_CLA,
		unsigned int mem_size_pr, unsigned int mem_size_CL,
		int nstates, int nRateCats, int nchar, int ncharGPU, dim3 dimBlock, dim3 dimGrid) {
	// copy matrices to the device
	CUDA_SAFE_CALL(hipMemcpy(d_Lpr, h_Lpr, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Rpr, h_Rpr, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_LCL, h_LCL, mem_size_CL, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_RCL, h_RCL, mem_size_CL, hipMemcpyHostToDevice));

	// run kernel
	if (nstates == 4)
	GarliCLANucleotideNRate<<< dimGrid, dimBlock >>>(d_CLA, d_Lpr, d_Rpr, d_LCL, d_RCL, nRateCats);
	else if (nstates == 20)
	GarliCLAAminoAcidNRate<<< dimGrid, dimBlock >>>(d_CLA, d_Lpr, d_Rpr, d_LCL, d_RCL, nRateCats);
	else
	GarliCLACodonNRate<<< dimGrid, dimBlock >>>(d_CLA, d_Lpr, d_Rpr, d_LCL, d_RCL, nRateCats);

	//check if kernel execution generated and error
	CUT_CHECK_ERROR("Kernel execution failed");

	// calculate remaining chars
	FLOAT_TYPE L1, R1;
	FLOAT_TYPE *dest = h_CLA + nstates * nRateCats * ncharGPU;
	h_LCL += nstates * nRateCats * ncharGPU;
	h_RCL += nstates * nRateCats * ncharGPU;
	for(unsigned int i=ncharGPU;i<nchar;i++) {
		for(unsigned int rate=0;rate<nRateCats;rate++) {
			for(unsigned int from=0;from<nstates;from++) {
				L1 = R1 = 0;
				for(unsigned int to=0;to<nstates;to++) {
					L1 += h_Lpr[rate*nstates*nstates + from*nstates + to] * h_LCL[to];
					R1 += h_Rpr[rate*nstates*nstates + from*nstates + to] * h_RCL[to];
				}
				dest[from] = L1 * R1;
			}
			h_LCL += nstates;
			h_RCL += nstates;
			dest += nstates;
		}
	}

	// copy result back to host
	CUDA_SAFE_CALL(hipMemcpy(h_CLA, d_CLA, mem_size_CL, hipMemcpyDeviceToHost));
}

extern "C"
void CuComputeGPUDeriv(const FLOAT_TYPE* h_partial, const FLOAT_TYPE* h_CL1, const int* h_partial_underflow_mult,
		const int* h_CL1_underflow_mult, const FLOAT_TYPE* h_prmat, const FLOAT_TYPE* h_d1mat, const FLOAT_TYPE* h_d2mat,
		const FLOAT_TYPE* h_rateProb, const FLOAT_TYPE* h_freqs, const int* h_countit, const int* h_conStates,
		FLOAT_TYPE* h_Tots, FLOAT_TYPE* h_Tots_arr, int* h_nchar_boot_index,
		FLOAT_TYPE* d_partial, FLOAT_TYPE* d_CL1, int* d_partial_underflow_mult,
		int* d_CL1_underflow_mult, FLOAT_TYPE* d_prmat, FLOAT_TYPE* d_d1mat, FLOAT_TYPE* d_d2mat,
		FLOAT_TYPE* d_rateProb, FLOAT_TYPE* d_freqs, int* d_countit, int* d_conStates,
		FLOAT_TYPE* d_Tots, FLOAT_TYPE* d_Tots_arr, int* d_nchar_boot_index,
		unsigned int mem_size_pr, unsigned int mem_size_CL, unsigned int mem_size_int_char,
		unsigned int mem_size_rates, unsigned int mem_size_states, unsigned int mem_size_Tots,
		unsigned int mem_size_Tots_arr, unsigned int mem_size_nchar_boot_index, int lastConst,
		bool NoPinvInModel, FLOAT_TYPE prI,
		int nstates, int nRateCats, int nchar, int ncharGPU, dim3 dimBlock, dim3 dimGrid) {
	// copy matrices to the device
	CUDA_SAFE_CALL(hipMemcpy(d_partial, h_partial, mem_size_CL, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_CL1, h_CL1, mem_size_CL, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_partial_underflow_mult, h_partial_underflow_mult, mem_size_int_char, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_CL1_underflow_mult, h_CL1_underflow_mult, mem_size_int_char, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_prmat, h_prmat, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_d1mat, h_d1mat, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_d2mat, h_d2mat, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_rateProb, h_rateProb, mem_size_rates, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_freqs, h_freqs, mem_size_states, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_countit, h_countit, mem_size_int_char, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_conStates, h_conStates, mem_size_int_char, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_nchar_boot_index, h_nchar_boot_index, mem_size_nchar_boot_index, hipMemcpyHostToDevice));

	// run kernel
	if (nstates == 4)
	GarliDerivNucleotideNRate<<< dimGrid, dimBlock >>>(d_partial, d_CL1, d_partial_underflow_mult,
			d_CL1_underflow_mult, d_prmat, d_d1mat, d_d2mat,
			d_rateProb, d_freqs, d_countit, d_conStates, d_Tots_arr, d_nchar_boot_index,
			lastConst, NoPinvInModel, prI, nRateCats);
	else if (nstates == 20)
	GarliDerivAminoAcidNRate<<< dimGrid, dimBlock >>>(d_partial, d_CL1, d_partial_underflow_mult,
			d_CL1_underflow_mult, d_prmat, d_d1mat, d_d2mat,
			d_rateProb, d_freqs, d_countit, d_conStates, d_Tots_arr, d_nchar_boot_index,
			lastConst, NoPinvInModel, prI, nRateCats);
	else
	GarliDerivCodonNRate<<< dimGrid, dimBlock >>>(d_partial, d_CL1, d_partial_underflow_mult,
			d_CL1_underflow_mult, d_prmat, d_d1mat, d_d2mat,
			d_rateProb, d_freqs, d_countit, d_conStates, d_Tots_arr, d_nchar_boot_index,
			lastConst, NoPinvInModel, prI, nRateCats);

	//check if kernel execution generated and error
	CUT_CHECK_ERROR("Kernel execution failed");

	// calculate remaining chars
	FLOAT_TYPE tot1=0, tot2=0, totL = 0;

	FLOAT_TYPE siteL, siteD1, siteD2;
	FLOAT_TYPE tempL, tempD1, tempD2;
	FLOAT_TYPE rateL, rateD1, rateD2;

	h_partial += nstates * nRateCats * ncharGPU;
	h_CL1 += nstates * nRateCats * ncharGPU;

	for(int i=ncharGPU;i<nchar;i++) {
		siteL = siteD1 = siteD2 = 0;
		for(int rate=0;rate<nRateCats;rate++) {
			rateL = rateD1 = rateD2 = 0;
			int rateOffset = rate*nstates*nstates;
			for(int from=0;from<nstates;from++) {
				tempL = tempD1 = tempD2 = 0;
				int offset = from * nstates;
				for(int to=0;to<nstates;to++) {
					tempL += h_prmat[rateOffset + offset + to]*h_CL1[to];
					tempD1 += h_d1mat[rateOffset + offset + to]*h_CL1[to];
					tempD2 += h_d2mat[rateOffset + offset + to]*h_CL1[to];
				}
				rateL += tempL * h_partial[from] * h_freqs[from];
				rateD1 += tempD1 * h_partial[from] * h_freqs[from];
				rateD2 += tempD2 * h_partial[from] * h_freqs[from];
			}
			siteL += rateL * h_rateProb[rate];
			siteD1 += rateD1 * h_rateProb[rate];
			siteD2 += rateD2 * h_rateProb[rate];
			h_partial += nstates;
			h_CL1 += nstates;
		}

		if((NoPinvInModel == false) && (h_nchar_boot_index[i]<=lastConst)) {
//			if (nstates == 4) {
//				float btot = 0.0f;
//				if (h_conStates[h_nchar_boot_index[i]] & 1)
//				btot += h_freqs[0];
//				if (h_conStates[h_nchar_boot_index[i]] & 2)
//				btot += h_freqs[1];
//				if (h_conStates[h_nchar_boot_index[i]] & 4)
//				btot += h_freqs[2];
//				if (h_conStates[h_nchar_boot_index[i]] & 8)
//				btot += h_freqs[3];
//				siteL += (prI * btot) * exp(h_partial_underflow_mult[h_nchar_boot_index[i]]
//						+ h_CL1_underflow_mult[h_nchar_boot_index[i]]);
//			} else
			siteL += (prI*h_freqs[h_conStates[h_nchar_boot_index[i]]] * exp((FLOAT_TYPE)h_partial_underflow_mult[h_nchar_boot_index[i]]) * exp((FLOAT_TYPE)h_CL1_underflow_mult[h_nchar_boot_index[i]]));
		}

		totL += (log(siteL) - h_partial_underflow_mult[h_nchar_boot_index[i]] - h_CL1_underflow_mult[h_nchar_boot_index[i]]) * h_countit[h_nchar_boot_index[i]];
		siteD1 /= siteL;
		tot1 += h_countit[h_nchar_boot_index[i]] * siteD1;
		tot2 += h_countit[h_nchar_boot_index[i]] * ((siteD2 / siteL) - siteD1*siteD1);
	}

	// copy result back to host
	CUDA_SAFE_CALL(hipMemcpy(h_Tots_arr, d_Tots_arr, mem_size_Tots_arr, hipMemcpyDeviceToHost));

	// clear previous results
	h_Tots[0] = 0;
	h_Tots[1] = 0;
	h_Tots[2] = 0;

	for (int i=0;i<dimGrid.x;i++) {
		h_Tots[0] += h_Tots_arr[i];
		h_Tots[1] += h_Tots_arr[i+dimGrid.x];
		h_Tots[2] += h_Tots_arr[i+dimGrid.x*2];
	}

	// add up the results from the remaining chars
	h_Tots[0] += totL;
	h_Tots[1] += tot1;
	h_Tots[2] += tot2;

}

extern "C"
void FreeGPU(FLOAT_TYPE* arr) {
	// free device memory
	CUDA_SAFE_CALL(hipFree(arr));
}

extern "C"
void FreePinnedMemory(void* arr) {
	// free host pinned memory
	CUDA_SAFE_CALL(hipHostFree(arr));

}

