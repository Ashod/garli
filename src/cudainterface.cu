#include "hip/hip_runtime.h"
/*
 * cudainterface.cu
 *
 *  Created on: Nov 12, 2008
 *      Author: ayres
 */

#include <iostream>

#include <cutil.h>

#include "defs.h"

// includes, kernels
#include "cudakernel.cu"

extern "C"
void AllocatePinnedMemory(void** arr, unsigned int mem_size_bytes) {
	// allocate host pinned memory
	CUDA_SAFE_CALL(hipHostMalloc((void**)&(*arr), mem_size_bytes));

}

extern "C"
void AllocateGPU(void** arr, unsigned int mem_size_bytes) {
	// allocate device memory
	CUDA_SAFE_CALL(hipMalloc((void**) &(*arr), mem_size_bytes));
}

extern "C"
void CuComputeGPUCLA(FLOAT_TYPE* h_Lpr, FLOAT_TYPE* h_Rpr, FLOAT_TYPE* h_LCL, FLOAT_TYPE* h_RCL, FLOAT_TYPE* h_CLA,
		FLOAT_TYPE* d_Lpr, FLOAT_TYPE* d_Rpr, FLOAT_TYPE* d_LCL, FLOAT_TYPE* d_RCL, FLOAT_TYPE* d_CLA,
		unsigned int mem_size_pr, unsigned int mem_size_CL,
		int nstates, int nRateCats, int nchar, int ncharGPU, dim3 dimBlock, dim3 dimGrid) {
	// copy matrices to the device
	CUDA_SAFE_CALL(hipMemcpy(d_Lpr, h_Lpr, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Rpr, h_Rpr, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_LCL, h_LCL, mem_size_CL, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_RCL, h_RCL, mem_size_CL, hipMemcpyHostToDevice));

	// run kernel
	if (nstates == 4)
	GarliCLANucleotideNRate<<< dimGrid, dimBlock >>>(d_CLA, d_Lpr, d_Rpr, d_LCL, d_RCL, nRateCats);
	else if (nstates == 20)
	GarliCLAAminoAcidNRate<<< dimGrid, dimBlock >>>(d_CLA, d_Lpr, d_Rpr, d_LCL, d_RCL, nRateCats);
	else
	GarliCLACodonNRate<<< dimGrid, dimBlock >>>(d_CLA, d_Lpr, d_Rpr, d_LCL, d_RCL, nRateCats);

	//check if kernel execution generated and error
	CUT_CHECK_ERROR("Kernel execution failed");

	// calculate remaining chars
	FLOAT_TYPE L1, R1;
	FLOAT_TYPE *dest = h_CLA + nstates * nRateCats * ncharGPU;
	h_LCL += nstates * nRateCats * ncharGPU;
	h_RCL += nstates * nRateCats * ncharGPU;
	for(unsigned int i=ncharGPU;i<nchar;i++) {
		for(unsigned int rate=0;rate<nRateCats;rate++) {
			for(unsigned int from=0;from<nstates;from++) {
				L1 = R1 = 0;
				for(unsigned int to=0;to<nstates;to++) {
					L1 += h_Lpr[rate*nstates*nstates + from*nstates + to] * h_LCL[to];
					R1 += h_Rpr[rate*nstates*nstates + from*nstates + to] * h_RCL[to];
				}
				dest[from] = L1 * R1;
			}
			h_LCL += nstates;
			h_RCL += nstates;
			dest += nstates;
		}
	}

	// copy result back to host
	CUDA_SAFE_CALL(hipMemcpy(h_CLA, d_CLA, mem_size_CL, hipMemcpyDeviceToHost));
}

extern "C"
void CuComputeGPUDeriv(const FLOAT_TYPE* h_partial, const FLOAT_TYPE* h_CL1, const int* h_partial_underflow_mult,
		const int* h_CL1_underflow_mult, const FLOAT_TYPE* h_prmat, const FLOAT_TYPE* h_d1mat, const FLOAT_TYPE* h_d2mat,
		const FLOAT_TYPE* h_rateProb, const FLOAT_TYPE* h_freqs, const int* h_countit, const int* h_conStates,
		FLOAT_TYPE* h_Tots, FLOAT_TYPE* h_Tots_arr, int* h_nchar_boot_index,
		FLOAT_TYPE* d_partial, FLOAT_TYPE* d_CL1, int* d_partial_underflow_mult,
		int* d_CL1_underflow_mult, FLOAT_TYPE* d_prmat, FLOAT_TYPE* d_d1mat, FLOAT_TYPE* d_d2mat,
		FLOAT_TYPE* d_rateProb, FLOAT_TYPE* d_freqs, int* d_countit, int* d_conStates,
		FLOAT_TYPE* d_Tots, FLOAT_TYPE* d_Tots_arr, int* d_nchar_boot_index,
		unsigned int mem_size_pr, unsigned int mem_size_CL, unsigned int mem_size_int_char,
		unsigned int mem_size_rates, unsigned int mem_size_states, unsigned int mem_size_Tots,
		unsigned int mem_size_Tots_arr, unsigned int mem_size_nchar_boot_index, int lastConst,
		bool NoPinvInModel, FLOAT_TYPE prI,
		int nstates, int nRateCats, int nchar, int ncharGPU, dim3 dimBlock, dim3 dimGrid) {

	// copy matrices to the device
	CUDA_SAFE_CALL(hipMemcpy(d_partial, h_partial, mem_size_CL, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_CL1, h_CL1, mem_size_CL, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_partial_underflow_mult, h_partial_underflow_mult, mem_size_int_char, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_CL1_underflow_mult, h_CL1_underflow_mult, mem_size_int_char, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_prmat, h_prmat, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_d1mat, h_d1mat, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_d2mat, h_d2mat, mem_size_pr, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_rateProb, h_rateProb, mem_size_rates, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_freqs, h_freqs, mem_size_states, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_countit, h_countit, mem_size_int_char, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_conStates, h_conStates, mem_size_int_char, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_nchar_boot_index, h_nchar_boot_index, mem_size_nchar_boot_index, hipMemcpyHostToDevice));

	// run kernel
	if (nstates == 4)
	GarliDerivNucleotideNRate<<< dimGrid, dimBlock >>>(d_partial, d_CL1, d_partial_underflow_mult,
			d_CL1_underflow_mult, d_prmat, d_d1mat, d_d2mat,
			d_rateProb, d_freqs, d_countit, d_conStates, d_Tots_arr, d_nchar_boot_index,
			lastConst, NoPinvInModel, prI, nRateCats);
	else if (nstates == 20)
	GarliDerivAminoAcidNRate<<< dimGrid, dimBlock >>>(d_partial, d_CL1, d_partial_underflow_mult,
			d_CL1_underflow_mult, d_prmat, d_d1mat, d_d2mat,
			d_rateProb, d_freqs, d_countit, d_conStates, d_Tots_arr, d_nchar_boot_index,
			lastConst, NoPinvInModel, prI, nRateCats);
	else
	GarliDerivCodonNRate<<< dimGrid, dimBlock >>>(d_partial, d_CL1, d_partial_underflow_mult,
			d_CL1_underflow_mult, d_prmat, d_d1mat, d_d2mat,
			d_rateProb, d_freqs, d_countit, d_conStates, d_Tots_arr, d_nchar_boot_index,
			lastConst, NoPinvInModel, prI, nRateCats);

	//check if kernel execution generated and error
	CUT_CHECK_ERROR("Kernel execution failed");

	// calculate remaining chars
	FLOAT_TYPE tot1=ZERO_POINT_ZERO, tot2=ZERO_POINT_ZERO, totL = ZERO_POINT_ZERO;

	FLOAT_TYPE siteL, siteD1, siteD2;
	FLOAT_TYPE tempL, tempD1, tempD2;
	FLOAT_TYPE rateL, rateD1, rateD2;

	h_partial += nstates * nRateCats * ncharGPU;
	h_CL1 += nstates * nRateCats * ncharGPU;

	for(int i=ncharGPU;i<nchar;i++) {
		siteL = siteD1 = siteD2 = ZERO_POINT_ZERO;
		for(int rate=0;rate<nRateCats;rate++) {
			rateL = rateD1 = rateD2 = ZERO_POINT_ZERO;
			int rateOffset = rate*nstates*nstates;
			for(int from=0;from<nstates;from++) {
				tempL = tempD1 = tempD2 = ZERO_POINT_ZERO;
				int offset = from * nstates;
				for(int to=0;to<nstates;to++) {
					tempL += h_prmat[rateOffset + offset + to]*h_CL1[to];
					tempD1 += h_d1mat[rateOffset + offset + to]*h_CL1[to];
					tempD2 += h_d2mat[rateOffset + offset + to]*h_CL1[to];
				}
				rateL += tempL * h_partial[from] * h_freqs[from];
				rateD1 += tempD1 * h_partial[from] * h_freqs[from];
				rateD2 += tempD2 * h_partial[from] * h_freqs[from];
			}
			siteL += rateL * h_rateProb[rate];
			siteD1 += rateD1 * h_rateProb[rate];
			siteD2 += rateD2 * h_rateProb[rate];
			h_partial += nstates;
			h_CL1 += nstates;
		}

		if((NoPinvInModel == false) && (h_nchar_boot_index[i]<=lastConst)) {
//			if (nstates == 4) {
//				float btot = 0.0f;
//				if (h_conStates[h_nchar_boot_index[i]] & 1)
//				btot += h_freqs[0];
//				if (h_conStates[h_nchar_boot_index[i]] & 2)
//				btot += h_freqs[1];
//				if (h_conStates[h_nchar_boot_index[i]] & 4)
//				btot += h_freqs[2];
//				if (h_conStates[h_nchar_boot_index[i]] & 8)
//				btot += h_freqs[3];
//				siteL += (prI * btot) * exp(h_partial_underflow_mult[h_nchar_boot_index[i]]
//						+ h_CL1_underflow_mult[h_nchar_boot_index[i]]);
//			} else
			siteL += (prI*h_freqs[h_conStates[h_nchar_boot_index[i]]] * exp((FLOAT_TYPE)h_partial_underflow_mult[h_nchar_boot_index[i]]) * exp((FLOAT_TYPE)h_CL1_underflow_mult[h_nchar_boot_index[i]]));
		}

		totL += (log(siteL) - h_partial_underflow_mult[h_nchar_boot_index[i]] - h_CL1_underflow_mult[h_nchar_boot_index[i]]) * h_countit[h_nchar_boot_index[i]];
		siteD1 /= siteL;
		tot1 += h_countit[h_nchar_boot_index[i]] * siteD1;
		tot2 += h_countit[h_nchar_boot_index[i]] * ((siteD2 / siteL) - siteD1*siteD1);
	}

	// copy result back to host
	CUDA_SAFE_CALL(hipMemcpy(h_Tots_arr, d_Tots_arr, mem_size_Tots_arr, hipMemcpyDeviceToHost));

	// clear previous results
	h_Tots[0] = 0;
	h_Tots[1] = 0;
	h_Tots[2] = 0;

	for (int i=0;i<dimGrid.x;i++) {
		h_Tots[0] += h_Tots_arr[i];
		h_Tots[1] += h_Tots_arr[i+dimGrid.x];
		h_Tots[2] += h_Tots_arr[i+dimGrid.x*2];
	}

	// add up the results from the remaining chars
	h_Tots[0] += totL;
	h_Tots[1] += tot1;
	h_Tots[2] += tot2;

}

extern "C"
void FreeGPU(FLOAT_TYPE* arr) {
	// free device memory
	CUDA_SAFE_CALL(hipFree(arr));
}

extern "C"
void FreePinnedMemory(void* arr) {
	// free host pinned memory
	CUDA_SAFE_CALL(hipHostFree(arr));

}

